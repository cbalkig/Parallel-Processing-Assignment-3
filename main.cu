#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define  N           32
#define  NUM_BLOCKS  N * N / NUM_THREADS
#define  NUM_THREADS 256
#define  THRESHOLD   0.001
void printMatrix(char *id, double matrix[N * N]) {
    char *s = (char *) malloc(10000 * sizeof(char));
    sprintf(s, "%s\n", id);
    for (int row = 0; row < N; row++) {
        sprintf(s, "%s\t\t\t\t\t", s);
        for (int columns = 0; columns < N; columns++) {
            sprintf(s, "%s%.2f ", s, matrix[row * N + columns]);
        }
        sprintf(s, "%s\n", s);
    }
    printf("%s", s);
    free(s);
}

__global__ void calc(double *a, double *b){
    // printf("Grid Dimension: %d,%d,%d Block Dimension: %d,%d,%d\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    int i = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + threadIdx.x + threadIdx.y;
    // printf("I'm Block:%d,%d,%d Thread:%d,%d,%d I'll operated on %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, i);

    if (i % N == 0) {
        b[i] = a[i];
    } else if (i / N == 0 && i % N == N - 1) {
        b[i] = (a[i] + a[i - 1] + a[i + N]) / 3;
    } else if (i / N == 0) {
        b[i] = (a[i] + a[i + 1] + a[i - 1] + a[i + N]) / 4;
    } else if (i / N == N - 1 && i % N == N - 1) {
        b[i] = (a[i] + a[i - 1] + a[i - N]) / 3;
    } else if (i % N == N - 1) {
        b[i] = (a[i] + a[i - 1] + a[i + N] + a[i - N]) / 4;
    } else if (i / N == N - 1) {
        b[i] = (a[i] + a[i + 1] + a[i - 1] + a[i - N]) / 4;
    } else {
        b[i] = (a[i] + a[i + 1] + a[i - 1] + a[i + N] + a[i - N]) / 5;
    }

    __syncthreads();
}

int main() {
    // CUDA configs
    // We will have Grid Dimension: 4,1,1 Block Dimension: 256,1,1
    dim3 dimGrid(NUM_BLOCKS, 1, 1);
    dim3 dimBlock(NUM_THREADS, 1, 1);
    int size = N * N * sizeof(double);
    double *a, *b;

    char *log = (char *) malloc(200 * sizeof(char));
    int temperature;
    printf("Please enter the heat temperature:\n");
    scanf("%d", &temperature);
    printf("Temperature: %d\n", temperature);
    if (temperature < 0 || temperature > 9) {
        printf("Heat temperature should be between 0 and 9.\n");
        exit(-1);
    }

    double A[N * N], B[N * N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = 0;
        }
    }
    printMatrix("Initial Matrix", A);

    for (int i = 0; i < N; i++) {
        A[i * N] = temperature;
    }
    printMatrix("Heat Matrix", A);

    int iteration_count = 0;
    while (iteration_count >= 0) { 
        // Ready for CUDA
	hipMalloc(&a, size);
        hipMalloc(&b, size);

	hipMemcpy(a, A, size, hipMemcpyHostToDevice);
        calc <<<dimGrid, dimBlock>>> (a, b);
        hipMemcpy(B, b, size, hipMemcpyDeviceToHost);

        iteration_count++;

        log = (char *) malloc(200 * sizeof(char));
        sprintf(log, "Iteration: %d\nMatrix", iteration_count);
        printMatrix(log, B);

        double diff_sum = 0;
        int diff_present = 0;
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                int idx = i * N + j;
                if (j != 0 && B[idx] > 0 && A[idx] > 0) {
                    diff_sum += (B[idx] - A[idx]);
                    diff_present++;
                }

               A[idx] = B[idx];
            }
        }

        double threshold = ((double) diff_sum / diff_present);
        printf("Total changes in the iteration: %f, Threshold: %f, Changed item count: %d\n", diff_sum, threshold, diff_present);
        if (diff_present > 0 && threshold < THRESHOLD) {
            printf("No change. Enough iterations: %d\n", iteration_count);
            exit(0);
        }
    }

    hipFree(a);
    hipFree(b);
    free(log);
    return 0;
}
