#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define  N           32
#define  NUM_BLOCKS  N * N / NUM_THREADS
#define  NUM_THREADS 4

void printMatrix(char *id, int matrix[N * N]) {
    char *s = (char *) malloc(10000 * sizeof(char));
    sprintf(s, "%s\n", id);
    for (int row = 0; row < N; row++) {
        sprintf(s, "%s\t\t\t\t\t", s);
        for (int columns = 0; columns < N; columns++) {
            sprintf(s, "%s%d ", s, matrix[row * N + columns]);
        }
        sprintf(s, "%s\n", s);
    }
    printf("%s", s);
    free(s);
}

__global__ void calc(int *a, int *b) {
    //printf("Grid Dimension: %d,%d,%d Block Dimension: %d,%d,%d\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
    int i = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y + threadIdx.x + threadIdx.y;
    //printf("I'm Block:%d,%d,%d Thread:%d,%d,%d I'll operated on %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, i);

    if (i % N == 0) {
        b[i] = a[i];
    } else if (i / N == 0 && i % N == N - 1) {
        b[i] = ceil((double) (a[i] + a[i - 1] + a[i + N]) / 3);
    } else if (i / N == 0) {
        b[i] = ceil((double) (a[i] + a[i + 1] + a[i - 1] + a[i + N]) / 4);
    } else if (i / N == N - 1 && i % N == N - 1) {
        b[i] = ceil((double) (a[i] + a[i - 1] + a[i - N]) / 3);
    } else if (i % N == N - 1) {
        b[i] = ceil((double) (a[i] + a[i - 1] + a[i + N] + a[i - N]) / 4);
    } else if (i / N == N - 1) {
        b[i] = ceil((double) (a[i] + a[i + 1] + a[i - 1] + a[i - N]) / 4);
    } else {
        b[i] = ceil((double) (a[i] + a[i + 1] + a[i - 1] + a[i + N] + a[i - N]) / 5);
    }

    __syncthreads();
}

int main() {
    // CUDA configs
    // We will have Grid Dimension: 256,1,1 Block Dimension: 4,1,1
    dim3 dimGrid(NUM_BLOCKS, 1, 1);
    dim3 dimBlock(NUM_THREADS, 1, 1);
    int size = N * N * sizeof(int);
    int *a, *b;

    char *log = (char *) malloc(200 * sizeof(char));
    int temperature;
    printf("Please enter the heat temperature:\n");
    scanf("%d", &temperature);
    printf("Temperature: %d\n", temperature);
    if (temperature < 0 || temperature > 9) {
        printf("Heat temperature should be between 0 and 9.\n");
        exit(-1);
    }

    int A[N * N], B[N * N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = 0;
        }
    }
    printMatrix("Initial Matrix", A);

    for (int i = 0; i < N; i++) {
        A[i * N] = temperature;
    }
    printMatrix("Heat Matrix", A);

    int iteration_count = 0;
    while (iteration_count >= 0) {
        // Ready for CUDA
        hipMalloc(&a, size);
        hipMalloc(&b, size);

        hipMemcpy(a, A, size, hipMemcpyHostToDevice);
        calc <<<dimGrid, dimBlock>>>(a, b);
        hipMemcpy(B, b, size, hipMemcpyDeviceToHost);

        iteration_count++;

        log = (char *) malloc(200 * sizeof(char));
        sprintf(log, "Iteration: %d\nMatrix", iteration_count);
        printMatrix(log, B);

        int diff_sum = 0;
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                int idx = i * N + j;
                if (B[idx] != 0) {
                    diff_sum += (B[idx] - A[idx]);
                }

                A[idx] = B[idx];
            }
        }

        double threshold = ((double) diff_sum / (N));
        printf("Total changes in the iteration: %d, Threshold: %f\n", diff_sum, threshold);
        if (threshold < 0.1) {
            printf("No change. Enough iterations: %d\n", iteration_count);
            exit(0);
        }
    }

    hipFree(a);
    hipFree(b);
    free(log);
    return 0;
}
